#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <mma.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

using namespace nvcuda;

#define M 128
#define N 128
#define K 128
#define TILE 16

#define MAX_VAL 2.0f       // 输入上限
#define SCALE_TARGET 0.25f // 缩放目标区间上限

// CUDA Core 版本（float输入，验证用）
__global__ void cuda_core_gemm(const float *A, const float *B, float *C) {
    int row = blockIdx.y * TILE + threadIdx.y;
    int col = blockIdx.x * TILE + threadIdx.x;
    if (row >= M || col >= N) return;

    float sum = 0.0f;
    for (int i = 0; i < K; ++i) {
        sum += A[row * K + i] * B[i * N + col];
    }
    C[row * N + col] = sum;
}

// Tensor Core 实现（按 16x16 tile 执行）
__global__ void tensor_core_gemm(const half *A, const half *B, float *C) {
    int warpRow = blockIdx.y;
    int warpCol = blockIdx.x;

    if (warpRow * TILE >= M || warpCol * TILE >= N) return;

    wmma::fragment<wmma::matrix_a, TILE, TILE, TILE, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, TILE, TILE, TILE, half, wmma::col_major> b_frag;
    wmma::fragment<wmma::accumulator, TILE, TILE, TILE, float> acc_frag;
    wmma::fill_fragment(acc_frag, 0.0f);

    const half *tile_a = A + (warpRow * TILE * K);
    const half *tile_b = B + (warpCol * TILE);

    for (int i = 0; i < K; i += TILE) {
        wmma::load_matrix_sync(a_frag, tile_a + i, K);
        wmma::load_matrix_sync(b_frag, tile_b + i * N, N);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);
    }

    float *tile_c = C + warpRow * TILE * N + warpCol * TILE;
    wmma::store_matrix_sync(tile_c, acc_frag, N, wmma::mem_row_major);
}

void check(hipError_t result, const char *msg) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s (%s)\n", msg, hipGetErrorString(result));
        exit(EXIT_FAILURE);
    }
}

int main() {
    float A_orig[M * K], B_orig[K * N];
    half A_half[M * K], B_half[K * N];
    float C_cuda[M * N] = {0}, C_tensor[M * N] = {0};

    float scale_in = SCALE_TARGET / MAX_VAL;
    float scale_out = 1.0f / (scale_in * scale_in);

    // 初始化原始输入并生成缩放后的 half 数据
    srand(time(NULL));
    for (int i = 0; i < M * K; ++i) {
        A_orig[i] = (rand() / (float)RAND_MAX) * MAX_VAL;
        A_half[i] = __float2half(A_orig[i] * scale_in);
    }
    for (int i = 0; i < K * N; ++i) {
        B_orig[i] = (rand() / (float)RAND_MAX) * MAX_VAL;
        B_half[i] = __float2half(B_orig[i] * scale_in);
    }

    float *d_Af, *d_Bf, *d_C_cuda, *d_C_tensor;
    half *d_Ah, *d_Bh;
    check(hipMalloc(&d_Af, sizeof(A_orig)), "hipMalloc A_float");
    check(hipMalloc(&d_Bf, sizeof(B_orig)), "hipMalloc B_float");
    check(hipMalloc(&d_C_cuda, sizeof(C_cuda)), "hipMalloc C_cuda");

    check(hipMalloc(&d_Ah, sizeof(A_half)), "hipMalloc A_half");
    check(hipMalloc(&d_Bh, sizeof(B_half)), "hipMalloc B_half");
    check(hipMalloc(&d_C_tensor, sizeof(C_tensor)), "hipMalloc C_tensor");

    check(hipMemcpy(d_Af, A_orig, sizeof(A_orig), hipMemcpyHostToDevice), "Memcpy A_float");
    check(hipMemcpy(d_Bf, B_orig, sizeof(B_orig), hipMemcpyHostToDevice), "Memcpy B_float");
    check(hipMemcpy(d_Ah, A_half, sizeof(A_half), hipMemcpyHostToDevice), "Memcpy A_half");
    check(hipMemcpy(d_Bh, B_half, sizeof(B_half), hipMemcpyHostToDevice), "Memcpy B_half");

    // CUDA Core 执行并计时
    hipEvent_t start1, stop1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1);
    hipEventRecord(start1);
    dim3 threads1(16, 16);
    dim3 blocks1((N + 15) / 16, (M + 15) / 16);
    cuda_core_gemm<<<blocks1, threads1>>>(d_Af, d_Bf, d_C_cuda);
    hipEventRecord(stop1);
    hipEventSynchronize(stop1);
    float time_cuda = 0;
    hipEventElapsedTime(&time_cuda, start1, stop1);

    // Tensor Core 执行并计时
    hipEvent_t start2, stop2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2);
    hipEventRecord(start2);
    dim3 threads2(32);  // 一个 warp
    dim3 blocks2(N / 16, M / 16);
    tensor_core_gemm<<<blocks2, threads2>>>(d_Ah, d_Bh, d_C_tensor);
    hipEventRecord(stop2);
    hipEventSynchronize(stop2);
    float time_tensor = 0;
    hipEventElapsedTime(&time_tensor, start2, stop2);

    check(hipMemcpy(C_cuda, d_C_cuda, sizeof(C_cuda), hipMemcpyDeviceToHost), "Memcpy C_cuda");
    check(hipMemcpy(C_tensor, d_C_tensor, sizeof(C_tensor), hipMemcpyDeviceToHost), "Memcpy C_tensor");

    // 反缩放恢复原精度
    for (int i = 0; i < M * N; ++i)
        C_tensor[i] *= scale_out;

    // 结果一致性检查
    bool match = true;
    for (int i = 0; i < M * N; ++i) {
        float diff = fabs(C_cuda[i] - C_tensor[i]);
        if (diff > 1e-2) {
            printf("Mismatch at %d: CUDA=%.2f, Tensor=%.2f, Diff=%.5f\n",
                   i, C_cuda[i], C_tensor[i], diff);
            match = false;
        }
    }

    printf("\nResult Match: %s\n", match ? "YES" : "NO");
    printf("CUDA Core Time:   %.4f ms\n", time_cuda);
    printf("Tensor Core Time: %.4f ms\n", time_tensor);

    hipFree(d_Af); hipFree(d_Bf);
    hipFree(d_Ah); hipFree(d_Bh);
    hipFree(d_C_cuda); hipFree(d_C_tensor);
    return 0;
}
